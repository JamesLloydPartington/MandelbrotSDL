#include "hip/hip_runtime.h"
#include <SDL2/SDL.h>
#include <SDL2/SDL_image.h>
#include <SDL2/SDL_timer.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <thrust/complex.h>
#include <math.h>
#include <iostream>

#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif

SDL_Window* win;
SDL_Renderer* rend;

//Width and Height of window
const int WIDTH = 1024;
const int HEIGHT = 1024;

//Classify a coordinate with a magnitude larger than DIVERGE as diverged (will stop further itterations of that coordinate).
const double DIVERGE = 2;

//Define class which give coordinate information
class coordInfo
{
  public:
    double start_x;
    double end_x;
    double start_y;
    double end_y;

    double step_x;
    double step_y;

    //Step sizes of real and imaginary axis
    void Steps()
    {
      step_x = (end_x - start_x) / (WIDTH - 1);
      step_y = (end_y - start_y) / (HEIGHT - 1);
    }

    //returns x coordinate given the pixel value on the x axis
    double xValue(int i)
    {
      return(start_x + i * step_x);
    }

    //returns y coordinate given the pixel value on the y axis
    double yValue(int j)
    {
      return(start_y + j * step_y);
    }
};

class fracVals
{
  public:
    thrust::complex<double> c; //Coordinate on complex plane
    thrust::complex<double> I = thrust::complex<double>(0, 1);; // 1i
    thrust::complex<double> z; //z_nth iteration
    int n; //Number of iterations
    bool isDiverged; //False if not above DIVERGE

    //Here you can change the iterative formular (Mandelbrot is z = z * z + c)
    CUDA_CALLABLE_MEMBER void Itterate()
    {
      /* Mandelbrot function
      z = z * z + c;
      if(thrust::abs(z) > DIVERGE)
      {
        isDiverged = true;
      }
      n++;
      */

      ////////////////////////////////////////////Custom formula start
      if(n == 0)
      {
        z = c;
      }
      z = thrust::tan(thrust::pow(z, z)) + c; //tan(z^z) + c
      //z = z * z + c;
      if(thrust::abs(z) > DIVERGE) //Mathematically this is incorrect for this formula since tan is periodic (This formula just creates a nice looking fractal)
      {
        isDiverged = true;
      }
      n++;
      ////////////////////////////////////////////Custom formula end
    }

    //Colour is a [Red, Green, Blue] array
    void RGB(coordInfo Grid, int N, int* colour)
    {
      //ORIGINAL DIMENTION


      if(n == N) //Give colour depending on where the iteration z is on the complex plane (green for real, blue for imaginary)
      {
        colour[0] = 200; //Customise value to change colour
        colour[1] = (int)floor(255 * (z.real() - DIVERGE) / (2 * DIVERGE));
        colour[2] = (int)floor(255 * (z.imag() - DIVERGE) / (2 * DIVERGE));
      }
      else //Give colour depending on how quickly z diverged. Darker the red, the quicker it diverged
      {
        colour[0] = (int)floor(255 * log(n) / log(N));
        colour[1] = 50; //Customise value to change colour
        colour[2] = 50; //Customise value to change colour
      }
    }
};

////////////////////////////////////////////////////////////////////////////////

fracVals** init_Fractal(coordInfo Grid) //Create 2D array of fractal points
{
  double x, y;
  int i, j;

  fracVals** M;
  hipMallocManaged(&M, WIDTH * sizeof(fracVals*));

  for(i = 0; i < WIDTH; i++)
  {
    x = Grid.xValue(i);
    hipMallocManaged(&M[i], HEIGHT * sizeof(fracVals));
    for(j = 0; j < HEIGHT; j++)
    {
      y = Grid.yValue(j);
      M[i][j].c= thrust::complex<double>(x, y);
      M[i][j].z= thrust::complex<double>(0, 0);
      M[i][j].n= 0;
      M[i][j].isDiverged = false;
    }
  }
  return(M);
}

__global__
void itterateAll_Fractal(fracVals** M, int N, coordInfo Grid)
{
  int i, j, k;
  //int index_k = threadIdx.z;
  //int stride_k = blockDim.z;

  i = blockIdx.x * blockDim.x + threadIdx.x;
  j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < WIDTH && j < HEIGHT)
  {
    for(k = 0; k < N; k++)
    {
      M[i][j].Itterate();
      if(M[i][j].isDiverged == true) //Once diverged, move to next point
      {
        break;
      }
    }
  }
}

void draw_Fractal(fracVals** M, int N, coordInfo Grid)
{
  hipDeviceSynchronize();
  SDL_RenderClear(rend);
  int i, j;
  int* colour = (int*)calloc(3, sizeof(int));
  for(i = 0; i < WIDTH; i++)
  {
    for(j = 0; j < HEIGHT; j++)
    {
      M[i][j].RGB(Grid, N, colour);
      SDL_SetRenderDrawColor(rend, colour[0], colour[1], colour[2], 0xFF); //Draw pixel
      SDL_RenderDrawPoint(rend, i, j); //Draw pixel

    }
  }
  SDL_RenderPresent(rend);
  printf("Fractal made %d\n", N);

  SDL_Texture* target = SDL_GetRenderTarget(rend);
  SDL_SetRenderTarget(rend, NULL);
  //SDL_QueryTexture(NULL, NULL, NULL, &width, &height);
  SDL_Surface* surface = SDL_CreateRGBSurface(0, WIDTH, HEIGHT, 32, 0, 0, 0, 0);
  SDL_RenderReadPixels(rend, NULL, surface->format->format, surface->pixels, surface->pitch);
  SDL_SaveBMP(surface, "Fractal.bmp");
  SDL_FreeSurface(surface);
  SDL_SetRenderTarget(rend, target);
}

////////////////////////////////////////////////////////////////////////////////

void init_SDL()
{
  // retutns zero on success else non-zero
  if (SDL_Init(SDL_INIT_EVERYTHING) != 0) {
      printf("error initializing SDL: %s\n", SDL_GetError());
  }
  win = SDL_CreateWindow("Fractal", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);

  // triggers the program that controls
  // your graphics hardware and sets flags
  Uint32 render_flags = SDL_RENDERER_TARGETTEXTURE;

  // creates a renderer to render our images
  rend = SDL_CreateRenderer(win, -1, render_flags);
}

void close_SDL()
{
  // destroy renderer
  SDL_DestroyRenderer(rend);

  // destroy window
  SDL_DestroyWindow(win);

  // close SDL
  SDL_Quit();
}

//SDL event function
void events_SDL()
{
  coordInfo media_SDL(double, double, double, double, int);
  // controls annimation loop
  int close = 0;
  int N = 1000;
  bool OldFractal;

  bool RightButton = false;
  bool LeftButton = false;
  //bool NewState = true;
  int* RightClickPos = (int*)calloc(2, sizeof(int)); //[x, y] top right
  int* LeftClickPos = (int*)calloc(2, sizeof(int)); //[x, y] bottom left

  // annimation loop
  int x, y;
  Uint32 buttons;
  SDL_Event event;

  int historySize = 1000; //History size of zoom
  int historyIndex = 0;
  coordInfo* GridHistory = (coordInfo*)calloc(historySize, sizeof(coordInfo)); //Stores history of different zooms

  while (!close) //While SDL window is open
  {
    historyIndex = 0;
    GridHistory[historyIndex] = media_SDL(-DIVERGE, DIVERGE, -DIVERGE, DIVERGE, N);
    OldFractal = true;

    // Events management
    while (SDL_PollEvent(&event) || OldFractal)
    {
      switch (event.type)
      {
        case SDL_QUIT:
          // handling of close button
          close = 1;
          OldFractal = false;
          break;

        case SDL_KEYDOWN:
          // keyboard API for key pressed
          switch (event.key.keysym.scancode)
          {
            case SDL_SCANCODE_E: //Changer number of iterations
              printf("Enter number of itterations\n");
              scanf("%d", &N);
              GridHistory[historyIndex] = media_SDL(GridHistory[historyIndex].start_x, GridHistory[historyIndex].end_x, GridHistory[historyIndex].start_y, GridHistory[historyIndex].end_y, N);
              break;

            case SDL_SCANCODE_BACKSPACE: //Zoom out
              if(historyIndex > 0)
              {
                printf("Undone zoom\n");
                historyIndex--;
                GridHistory[historyIndex + 1] = media_SDL(GridHistory[historyIndex].start_x, GridHistory[historyIndex].end_x, GridHistory[historyIndex].start_y, GridHistory[historyIndex].end_y, N);
              }
              else
              {
                printf("Already at original zoom\n");
              }
              break;

            default:
              break;
          }
      }

      SDL_PumpEvents();  // make sure we have the latest mouse state.
      buttons = SDL_GetMouseState(&x, &y);

      if ((buttons & SDL_BUTTON_LMASK) != 0) //Bottom left (left click)
      {
        LeftButton = true;
        LeftClickPos[0] = x;
        LeftClickPos[1] = y;
      }

      if ((buttons & SDL_BUTTON_RMASK) != 0) //Top right (right click)
      {
        RightButton = true;
        RightClickPos[0] = x;
        RightClickPos[1] = y;
      }

      if(LeftButton && RightButton) //Once both left and right corners are made, zoom in.
      {
        printf("%f %f %f %f\n", GridHistory[historyIndex].xValue(LeftClickPos[0]), GridHistory[historyIndex].yValue(LeftClickPos[1]), GridHistory[historyIndex].xValue(RightClickPos[0]), GridHistory[historyIndex].yValue(RightClickPos[1]));

        GridHistory[historyIndex + 1] = media_SDL(GridHistory[historyIndex].xValue(LeftClickPos[0]), GridHistory[historyIndex].xValue(RightClickPos[0]), GridHistory[historyIndex].yValue(LeftClickPos[1]), GridHistory[historyIndex].yValue(RightClickPos[1]), N);
        historyIndex++;

        LeftButton = false;
        RightButton = false;
      }
    }
    SDL_DestroyRenderer(rend);
  }
}

coordInfo media_SDL(double start_x, double end_x, double start_y, double end_y, int N)
{
  fracVals** init_Fractal(coordInfo);

  coordInfo Grid;
  Grid.start_x = start_x;
  Grid.end_x = end_x;
  Grid.start_y = start_y;
  Grid.end_y = end_y;

  Grid.Steps();

  //Total threads = THREADS_x * THREADS_y, MAXIMUM IS 32 by 32 for this program on any GPU (even if it has more than 1024 threads).
  int THREADS_x = 32;
  int THREADS_y = 32;
  int BLOCKSIZE_x = (int)ceil(WIDTH / THREADS_x) + 1;
  int BLOCKSIZE_y = (int)ceil(HEIGHT / THREADS_y) + 1;

  dim3 threads(THREADS_x, THREADS_y);
  dim3 blocks(BLOCKSIZE_x, BLOCKSIZE_y);

  fracVals** Fractal = init_Fractal(Grid);
  itterateAll_Fractal<<<blocks, threads>>>(Fractal, N, Grid);

  void draw_Fractal(fracVals**, int, coordInfo);
  draw_Fractal(Fractal, N, Grid);

  return Grid;
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])
{
  //Start SDL
  init_SDL();

  //SDL events
  events_SDL();

  //Close SDL
  close_SDL();

  return 0;
}
